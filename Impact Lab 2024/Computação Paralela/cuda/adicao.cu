
#include <hip/hip_runtime.h>
#include <stdio.h>
#define N 512

__global__ void add(int *a, int *b, int *c){
    c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];
}

int main(void){

    int a, b, c;
    int *d_a, *d_b, *d_c;
    int size = N * sizeof(int);

    //Good Practice Naming - Memory Allocation
    // d_validable
    // h_valiable

    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);
    hipMalloc((void **)&d_c, size);

    a = 2;
    b = 7;

    //Copy inputs on device
    // cumdaMemcpy(Destination, Source, size, Directions);

    hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);

    //Launch add() kernel on GPU
    add<<<1,1>>>(d_a, d_b, d_c);

    //Copy result back to host
    hipMemcpy(&c, d_c, size, hipMemcpyDeviceToHost);

    //Cleanup
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;

}

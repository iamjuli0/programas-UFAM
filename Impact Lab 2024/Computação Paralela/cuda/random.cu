
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define N 12

void random_ints(int *array, int size) {
    for (int i = 0; i < size; i++) {
        array[i] = rand() % 100; // Gera números aleatórios entre 0 e 99
    }
}

__global__ void add(int *a, int *b, int *c) {
    int index = threadIdx.x;
    if (index < N) {
        c[index] = a[index] + b[index];
    }
}

int main(void) {
    int *a, *b, *c;
    int *d_a, *d_b, *d_c;
    int size = N * sizeof(int);

    // Inicializa a semente para números aleatórios
    srand(time(NULL));

    // Alocação de memória no host
    a = (int *)malloc(size);
    random_ints(a, N);
    b = (int *)malloc(size);
    random_ints(b, N);
    c = (int *)malloc(size);

    // Alocação de memória no dispositivo
    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);
    hipMalloc((void **)&d_c, size);

    // Cópia dos dados do host para o dispositivo
    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    // Chamada do kernel com N threads
    add<<<1, N>>>(d_a, d_b, d_c);

    // Cópia dos resultados do dispositivo para o host
    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    // Exibir os resultados
    for (int i = 0; i < N; i++) {
        printf("%d + %d = %d\n", a[i], b[i], c[i]);
    }

    // Liberação da memória
    free(a);
    free(b);
    free(c);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}